#include "hip/hip_runtime.h"
#include "voxelize.cuh"

// CUDA Global Memory variables

// Debug counters for some sanity checks
#ifdef _DEBUG
__device__ size_t debug_d_n_voxels_marked = 0;
__device__ size_t debug_d_n_triangles = 0;
__device__ size_t debug_d_n_voxels_tested = 0;
#endif

// Possible optimization: buffer bitsets (for now: Disabled because too much overhead)
//struct bufferedBitSetter{
//	unsigned int* voxel_table;
//	size_t current_int_location;
//	unsigned int current_mask;
//
//	__device__ __inline__ bufferedBitSetter(unsigned int* voxel_table, size_t index) :
//		voxel_table(voxel_table), current_mask(0) {
//		current_int_location = int(index / 32.0f);
//	}
//
//	__device__ __inline__ void setBit(size_t index){
//		size_t new_int_location = int(index / 32.0f);
//		if (current_int_location != new_int_location){
//			flush();
//			current_int_location = new_int_location;
//		}
//		unsigned int bit_pos = 31 - (unsigned int)(int(index) % 32);
//		current_mask = current_mask | (1 << bit_pos);
//	}
//
//	__device__ __inline__ void flush(){
//		if (current_mask != 0){
//			atomicOr(&(voxel_table[current_int_location]), current_mask);
//		}
//	}
//};

// Possible optimization: check bit before you set it - don't need to do atomic operation if it's already set to 1
// For now: overhead, so it seems
//__device__ __inline__ bool checkBit(unsigned int* voxel_table, size_t index){
//	size_t int_location = index / size_t(32);
//	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
//	return ((voxel_table[int_location]) & (1 << bit_pos));
//}

// readablesizestrings
//__device__ inline std::string readableSize(size_t bytes) {
//	double bytes_d = static_cast<double>(bytes);
//	std::string r;
//	if (bytes_d <= 0) r = "0 Bytes";
//	else if (bytes_d >= 1099511627776.0) r = std::to_string(static_cast<size_t>(bytes_d / 1099511627776.0)) + " TB";
//	else if (bytes_d >= 1073741824.0) r = std::to_string(static_cast<size_t>(bytes_d / 1073741824.0)) + " GB";
//	else if (bytes_d >= 1048576.0) r = std::to_string(static_cast<size_t>(bytes_d / 1048576.0)) + " MB";
//	else if (bytes_d >= 1024.0) r = std::to_string(static_cast<size_t>(bytes_d / 1024.0)) + " KB";
//	else r = std::to_string(static_cast<size_t>(bytes_d)) + " bytes";
//	return r;
// };

// Set a bit in the giant voxel table. This involves doing an atomic operation on a 32-bit word in memory.
// Blocking other threads writing to it for a very short time
__device__ __inline__ void setBit(unsigned int* voxel_table, size_t index){
	size_t int_location = index / size_t(32);
	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
	unsigned int mask = 1 << bit_pos;
	atomicOr(&(voxel_table[int_location]), mask);
}

__device__ bool SameSideTri(glm::vec3 v1,glm::vec3 v2,glm::vec3 v3,glm::vec3 X,glm::vec3 p)
{
  // Edge vectors
  glm::vec3 e0 = v2 - v1;
  glm::vec3 e1 = v3 - v2;
  glm::vec3 e2 = v1 - v3;
  // Normal vector pointing up from the triangle
  glm::vec3 n = glm::normalize(glm::cross(e0, e1));
  //glm::vec3 normal = glm::cross(v2 - v1, v3 - v1);
  float dotX = glm::dot(n, X - v1);
  float dotP = glm::dot(n, p - v1);
  if (signbit(dotX)==signbit(dotP)||dotP==0){
    return true;
    }
  else
    {
    return false;
    }
}
  // check if the point P is inside the tetrahedron (v1,v2,v3,v4)
__device__ bool PointInTetrahedron(glm::vec3 v1, glm::vec3 v2,glm::vec3 v3,glm::vec3 v4,glm::vec3 p)
{
  return (SameSideTri(v1, v2, v3, v4, p) &&
	  SameSideTri(v2, v3, v4, v1, p) &&
	  SameSideTri(v3, v4, v1, v2, p) &&
	  SameSideTri(v4, v1, v2, v3, p));
}


// Main triangle voxelization method
__global__ void voxelize_triangle(voxinfo info, float* triangle_data, long* greyscale_data, unsigned int* voxel_table, unsigned short* result){
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	// Common variables used in the voxelization process
	glm::vec3 delta_p(info.unit.x, info.unit.y, info.unit.z);
	glm::vec3 grid_max(info.gridsize.x - 1, info.gridsize.y - 1, info.gridsize.z - 1); // grid max (grid runs from 0 to gridsize-1)

	while (thread_id < info.n_triangles){ // every thread works on specific triangles in its stride
		size_t t = thread_id * 9; // triangle contains 9 vertices
        size_t tri_index = size_t(t/9);
		// COMPUTE COMMON TRIANGLE PROPERTIES
		// Move vertices to origin using bbox
		glm::vec3 v0 = glm::vec3(triangle_data[t], triangle_data[t + 1], triangle_data[t + 2]) - info.bbox.min;
		glm::vec3 v1 = glm::vec3(triangle_data[t + 3], triangle_data[t + 4], triangle_data[t + 5]) - info.bbox.min; 
		glm::vec3 v2 = glm::vec3(triangle_data[t + 6], triangle_data[t + 7], triangle_data[t + 8]) - info.bbox.min;
		// Edge vectors
		glm::vec3 e0 = v1 - v0;
		glm::vec3 e1 = v2 - v1;
		glm::vec3 e2 = v0 - v2;
		// Normal vector pointing up from the triangle
		glm::vec3 n = glm::normalize(glm::cross(e0, e1));

		// COMPUTE TRIANGLE BBOX IN GRID
		// Triangle bounding box in world coordinates is min(v0,v1,v2) and max(v0,v1,v2)
		AABox<glm::vec3> t_bbox_world(glm::min(v0, glm::min(v1, v2)), glm::max(v0, glm::max(v1, v2)));
		// Triangle bounding box in voxel grid coordinates is the world bounding box divided by the grid unit vector
		AABox<glm::ivec3> t_bbox_grid;
		t_bbox_grid.min = glm::clamp(t_bbox_world.min / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);
		t_bbox_grid.max = glm::clamp(t_bbox_world.max / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);

		// PREPARE PLANE TEST PROPERTIES
		glm::vec3 c(0.0f, 0.0f, 0.0f);
		if (n.x > 0.0f) { c.x = info.unit.x; }
		if (n.y > 0.0f) { c.y = info.unit.y; }
		if (n.z > 0.0f) { c.z = info.unit.z; }
		float d1 = glm::dot(n, (c - v0));
		float d2 = glm::dot(n, ((delta_p - c) - v0));

		// PREPARE PROJECTION TEST PROPERTIES
		// XY plane
		glm::vec2 n_xy_e0(-1.0f*e0.y, e0.x);
		glm::vec2 n_xy_e1(-1.0f*e1.y, e1.x);
		glm::vec2 n_xy_e2(-1.0f*e2.y, e2.x);
		if (n.z < 0.0f) {
			n_xy_e0 = -n_xy_e0;
			n_xy_e1 = -n_xy_e1;
			n_xy_e2 = -n_xy_e2;
		}
		float d_xy_e0 = (-1.0f * glm::dot(n_xy_e0, glm::vec2(v0.x, v0.y))) + glm::max(0.0f, info.unit.x*n_xy_e0[0]) + glm::max(0.0f, info.unit.y*n_xy_e0[1]);
		float d_xy_e1 = (-1.0f * glm::dot(n_xy_e1, glm::vec2(v1.x, v1.y))) + glm::max(0.0f, info.unit.x*n_xy_e1[0]) + glm::max(0.0f, info.unit.y*n_xy_e1[1]);
		float d_xy_e2 = (-1.0f * glm::dot(n_xy_e2, glm::vec2(v2.x, v2.y))) + glm::max(0.0f, info.unit.x*n_xy_e2[0]) + glm::max(0.0f, info.unit.y*n_xy_e2[1]);
		// YZ plane
		glm::vec2 n_yz_e0(-1.0f*e0.z, e0.y);
		glm::vec2 n_yz_e1(-1.0f*e1.z, e1.y);
		glm::vec2 n_yz_e2(-1.0f*e2.z, e2.y);
		if (n.x < 0.0f) {
			n_yz_e0 = -n_yz_e0;
			n_yz_e1 = -n_yz_e1;
			n_yz_e2 = -n_yz_e2;
		}
		float d_yz_e0 = (-1.0f * glm::dot(n_yz_e0, glm::vec2(v0.y, v0.z))) + glm::max(0.0f, info.unit.y*n_yz_e0[0]) + glm::max(0.0f, info.unit.z*n_yz_e0[1]);
		float d_yz_e1 = (-1.0f * glm::dot(n_yz_e1, glm::vec2(v1.y, v1.z))) + glm::max(0.0f, info.unit.y*n_yz_e1[0]) + glm::max(0.0f, info.unit.z*n_yz_e1[1]);
		float d_yz_e2 = (-1.0f * glm::dot(n_yz_e2, glm::vec2(v2.y, v2.z))) + glm::max(0.0f, info.unit.y*n_yz_e2[0]) + glm::max(0.0f, info.unit.z*n_yz_e2[1]);
		// ZX plane
		glm::vec2 n_zx_e0(-1.0f*e0.x, e0.z);
		glm::vec2 n_zx_e1(-1.0f*e1.x, e1.z);
		glm::vec2 n_zx_e2(-1.0f*e2.x, e2.z);
		if (n.y < 0.0f) {
			n_zx_e0 = -n_zx_e0;
			n_zx_e1 = -n_zx_e1;
			n_zx_e2 = -n_zx_e2;
		}
		float d_xz_e0 = (-1.0f * glm::dot(n_zx_e0, glm::vec2(v0.z, v0.x))) + glm::max(0.0f, info.unit.x*n_zx_e0[0]) + glm::max(0.0f, info.unit.z*n_zx_e0[1]);
		float d_xz_e1 = (-1.0f * glm::dot(n_zx_e1, glm::vec2(v1.z, v1.x))) + glm::max(0.0f, info.unit.x*n_zx_e1[0]) + glm::max(0.0f, info.unit.z*n_zx_e1[1]);
		float d_xz_e2 = (-1.0f * glm::dot(n_zx_e2, glm::vec2(v2.z, v2.x))) + glm::max(0.0f, info.unit.x*n_zx_e2[0]) + glm::max(0.0f, info.unit.z*n_zx_e2[1]);

		// test possible grid boxes for overlap
		for (int z = t_bbox_grid.min.z; z <= t_bbox_grid.max.z; z++){
			for (int y = t_bbox_grid.min.y; y <= t_bbox_grid.max.y; y++){
				for (int x = t_bbox_grid.min.x; x <= t_bbox_grid.max.x; x++){
					// size_t location = x + (y*info.gridsize) + (z*info.gridsize*info.gridsize);
					// if (checkBit(voxel_table, location)){ continue; }
#ifdef _DEBUG
					atomicAdd(&debug_d_n_voxels_tested, 1);
#endif
					// TRIANGLE PLANE THROUGH BOX TEST
					glm::vec3 p(x*info.unit.x, y*info.unit.y, z*info.unit.z);
					float nDOTp = glm::dot(n, p);
					if ((nDOTp + d1) * (nDOTp + d2) > 0.0f) { continue; }

					// PROJECTION TESTS
					// XY
					glm::vec2 p_xy(p.x, p.y);
					if ((glm::dot(n_xy_e0, p_xy) + d_xy_e0) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e1, p_xy) + d_xy_e1) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e2, p_xy) + d_xy_e2) < 0.0f){ continue; }

					// YZ
					glm::vec2 p_yz(p.y, p.z);
					if ((glm::dot(n_yz_e0, p_yz) + d_yz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e1, p_yz) + d_yz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e2, p_yz) + d_yz_e2) < 0.0f){ continue; }

					// XZ	
					glm::vec2 p_zx(p.z, p.x);
					if ((glm::dot(n_zx_e0, p_zx) + d_xz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e1, p_zx) + d_xz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e2, p_zx) + d_xz_e2) < 0.0f){ continue; }

#ifdef _DEBUG
					atomicAdd(&debug_d_n_voxels_marked, 1);
#endif

						size_t location = static_cast<size_t>(x) + (static_cast<size_t>(y)* static_cast<size_t>(info.gridsize.y)) + (static_cast<size_t>(z)* static_cast<size_t>(info.gridsize.y)* static_cast<size_t>(info.gridsize.z));
						setBit(voxel_table, location);
                        result[location] = static_cast<unsigned short>(greyscale_data[tri_index]);
					continue;
				}
			}
		}
#ifdef _DEBUG
		atomicAdd(&debug_d_n_triangles, 1);
#endif
		thread_id += stride;
	}
}

// Main tetrahedron voxelization method
__global__ void voxelize_tetra(voxinfo info, float* tet_data, long* greyscale_data, unsigned int* voxel_table, unsigned short* result){
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	// Common variables used in the voxelization process
	glm::vec3 delta_p(info.unit.x, info.unit.y, info.unit.z);
	glm::vec3 grid_max(info.gridsize.x - 1, info.gridsize.y - 1, info.gridsize.z - 1); // grid max (grid runs from 0 to gridsize-1)

	while (thread_id < info.n_triangles){ // every thread works on specific tetrahedron in its stride
		size_t t = thread_id * 12; // tetrahedron contains 12 vertices
        size_t tri_index = size_t(t/12);
		// COMPUTE COMMON TET PROPERTIES
		// Move vertices to origin using bbox
		glm::vec3 A = glm::vec3(tet_data[t], tet_data[t + 1], tet_data[t + 2]) - info.bbox.min;
		glm::vec3 B = glm::vec3(tet_data[t + 3], tet_data[t + 4], tet_data[t + 5]) - info.bbox.min; 
		glm::vec3 C = glm::vec3(tet_data[t + 6], tet_data[t + 7], tet_data[t + 8]) - info.bbox.min;
        glm::vec3 D = glm::vec3(tet_data[t + 9], tet_data[t + 10], tet_data[t + 11]) - info.bbox.min;
		

		// COMPUTE TETRA BBOX IN GRID
		// Tetrahedron bounding box in world coordinates is min(A,B,C,D) and max(A,B,C,D)
        AABox<glm::vec3> t_bbox_world(glm::min(A, glm::min(B, glm::min(C,D))),glm::max(A, glm::max(B, glm::max(C,D))));
		// Tetrahedron bounding box in voxel grid coordinates is the world bounding box divided by the grid unit vector
		AABox<glm::ivec3> t_bbox_grid;
		t_bbox_grid.min = glm::clamp(t_bbox_world.min / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);
		t_bbox_grid.max = glm::clamp(t_bbox_world.max / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);

		
		// test possible grid boxes for overlap
		for (int z = t_bbox_grid.min.z; z <= t_bbox_grid.max.z; z++){
			for (int y = t_bbox_grid.min.y; y <= t_bbox_grid.max.y; y++){
				for (int x = t_bbox_grid.min.x; x <= t_bbox_grid.max.x; x++){
					// size_t location = x + (y*info.gridsize) + (z*info.gridsize*info.gridsize);
					// if (checkBit(voxel_table, location)){ continue; }
                    glm::vec3 P =  glm::vec3((x+0.5)*info.unit.x,(y+0.5)*info.unit.y,(z+0.5)*info.unit.z);
			        // check if point p is on the "correct" side of all 4 triangles and thus inside the tetrahedron.
			        if(PointInTetrahedron(A,B,C,D,P))
                    {
						size_t location = static_cast<size_t>(x) + (static_cast<size_t>(y)* static_cast<size_t>(info.gridsize.y)) + (static_cast<size_t>(z)* static_cast<size_t>(info.gridsize.y)* static_cast<size_t>(info.gridsize.z));
						setBit(voxel_table, location);
                        result[location] = static_cast<unsigned short>(greyscale_data[tri_index]);
					}
					continue;
				    }
			    }
		    }
		thread_id += stride;
	    }
}

void voxelize(const voxinfo& v, float* element_data, long* greyscale_data, unsigned int* vtable, bool useThrustPath, unsigned short* result_array, bool use_tetra) {
	float elapsedTime;

	// These are only used when we're not using UNIFIED memory
	unsigned int* dev_vtable; // DEVICE pointer to voxel_data
    unsigned short* dev_result; // DEVICE pointer to voxel_data
	size_t vtable_size; // vtable size
	size_t result_size; // result size

	// Create timers, set start time
	hipEvent_t start_vox, stop_vox;
	checkCudaErrors(hipEventCreate(&start_vox));
	checkCudaErrors(hipEventCreate(&stop_vox));

	// Estimate best block and grid size using CUDA Occupancy Calculator
	int blockSize;   // The launch configurator returned block size 
	int minGridSize; // The minimum grid size needed to achieve the  maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxelize_triangle, 0, 0);
	// Round up according to array size 
	gridSize = (v.n_triangles + blockSize - 1) / blockSize;

	if (useThrustPath) { // We're not using UNIFIED memory
		vtable_size = ((size_t)v.gridsize.x * v.gridsize.y * v.gridsize.z) / (size_t) 8.0;
        result_size = ((size_t)v.gridsize.x * v.gridsize.y * v.gridsize.z)*sizeof(unsigned short);

		fprintf(stdout, "[Voxel Grid] Allocating %s of DEVICE memory for Voxel Grid\n", readableSize(vtable_size).c_str());
		checkCudaErrors(hipMalloc(&dev_vtable, vtable_size));
		checkCudaErrors(hipMemset(dev_vtable, 0, vtable_size));

        fprintf(stdout, "[Voxel Grid] Allocating %s of DEVICE memory for Result\n", readableSize(result_size).c_str());
        checkCudaErrors(hipMalloc(&dev_result, result_size));
		checkCudaErrors(hipMemset(dev_result, 0, result_size));

		// Start voxelization
		checkCudaErrors(hipEventRecord(start_vox, 0));
    if (use_tetra){voxelize_tetra << <gridSize, blockSize >> > (v, element_data, greyscale_data, dev_vtable, dev_result);}
	else {voxelize_triangle << <gridSize, blockSize >> > (v, element_data, greyscale_data, dev_vtable, dev_result);}
	}
	else { // UNIFIED MEMORY 
		checkCudaErrors(hipEventRecord(start_vox, 0));
    if (use_tetra){voxelize_tetra << <gridSize, blockSize >> > (v, element_data, greyscale_data, vtable, result_array);}
	else {voxelize_triangle << <gridSize, blockSize >> > (v, element_data, greyscale_data, vtable, result_array);}
	}

	hipDeviceSynchronize();
	checkCudaErrors(hipEventRecord(stop_vox, 0));
	checkCudaErrors(hipEventSynchronize(stop_vox));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start_vox, stop_vox));
	printf("[Perf] Voxelization GPU time: %.1f ms\n", elapsedTime);

	// If we're not using UNIFIED memory, copy the voxel table back and free all
	if (useThrustPath){
		fprintf(stdout, "[Voxel Grid] Copying %s to page-locked HOST memory\n", readableSize(vtable_size).c_str());
		checkCudaErrors(hipMemcpy((void*)vtable, dev_vtable, vtable_size, hipMemcpyDefault));
        fprintf(stdout, "[Voxel Grid] Copying %s to page-locked HOST memory\n", readableSize(result_size).c_str());
		checkCudaErrors(hipMemcpy((void*)result_array, dev_result, result_size, hipMemcpyDefault));
		fprintf(stdout, "[Voxel Grid] Freeing %s of DEVICE memory\n", readableSize(vtable_size).c_str());
		checkCudaErrors(hipFree(dev_vtable));
        fprintf(stdout, "[Voxel Grid] Freeing %s of DEVICE memory\n", readableSize(result_size).c_str());
		checkCudaErrors(hipFree(dev_result));
	}

	// SANITY CHECKS
#ifdef _DEBUG
	size_t debug_n_triangles, debug_n_voxels_marked, debug_n_voxels_tested;
	checkCudaErrors(hipMemcpyFromSymbol((void*)&(debug_n_triangles),debug_d_n_triangles, sizeof(debug_d_n_triangles), 0, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol((void*)&(debug_n_voxels_marked), debug_d_n_voxels_marked, sizeof(debug_d_n_voxels_marked), 0, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol((void*) & (debug_n_voxels_tested), debug_d_n_voxels_tested, sizeof(debug_d_n_voxels_tested), 0, hipMemcpyDeviceToHost));
	printf("[Debug] Processed %zu triangles on the GPU \n", debug_n_triangles);
	printf("[Debug] Tested %zu voxels for overlap on GPU \n", debug_n_voxels_tested);
	printf("[Debug] Marked %zu voxels as filled (includes duplicates!) \n", debug_n_voxels_marked);
#endif

	// Destroy timers
	checkCudaErrors(hipEventDestroy(start_vox));
	checkCudaErrors(hipEventDestroy(stop_vox));
}
